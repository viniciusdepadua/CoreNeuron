#include "hip/hip_runtime.h"
/*
# =============================================================================
# Copyright (c) 2016 - 2021 Blue Brain Project/EPFL
#
# See top-level LICENSE file for details.
# =============================================================================.
*/
#include "coreneuron/mpi/core/nrnmpi.hpp"
#include "coreneuron/utils/memory.h"
#include "coreneuron/utils/nrnmutdec.hpp"
#include "coreneuron/utils/randoms/nrnran123.h"

#include <cmath>
#include <iostream>
#include <memory>
#include <mutex>

// In a GPU build this file will be compiled by NVCC as CUDA code
// In a CPU build this file will be compiled by a C++ compiler as C++ code
#ifdef __HIPCC__
#define CORENRN_DEVICE __device__
#else
#define CORENRN_DEVICE
#endif

namespace {
/* Global data structure per process. Using a unique_ptr here causes [minor]
 * problems because its destructor can be called very late during application
 * shutdown. If the destructor calls hipFree and the CUDA runtime has already
 * been shut down then tools like cuda-memcheck reports errors.
 */
philox4x32_key_t* g_k{};

// In a GPU build we need a device-side global pointer to this global state.
// This is set to the same unified memory address as `g_k` in
// `setup_global_state()` if the GPU is enabled. It would be cleaner to use
// __managed__ here, but unfortunately that does not work on machines that do
// not have a GPU.
#ifdef __HIPCC__
CORENRN_DEVICE philox4x32_key_t* g_k_dev;
#endif

OMP_Mutex g_instance_count_mutex;
std::size_t g_instance_count{};

constexpr double SHIFT32 = 1.0 / 4294967297.0; /* 1/(2^32 + 1) */

void setup_global_state() {
    if (g_k) {
        // Already initialised, nothing to do
        return;
    }
    g_k = coreneuron::allocate_unique<philox4x32_key_t>(
              coreneuron::unified_allocator<philox4x32_key_t>{})
              .release();
#ifdef __HIPCC__
    if (coreneuron::unified_memory_enabled()) {
        // Set the device-side global g_k_dev to point to the newly-allocated
        // unified memory. If this is false, g_k is just a host pointer and
        // there is no point initialising the device global to it.
        {
            auto const code = hipMemcpyToSymbol(HIP_SYMBOL(g_k_dev), &g_k, sizeof(g_k));
            assert(code == hipSuccess);
        }
        // Make sure g_k_dev is updated.
        {
            auto const code = hipDeviceSynchronize();
            assert(code == hipSuccess);
        }
    }
#endif
}

/** @brief Get the Random123 global state from either host or device code.
 */
CORENRN_HOST_DEVICE philox4x32_key_t& get_global_state() {
    philox4x32_key_t* ret{nullptr};
#ifdef __CUDA_ARCH__
    // Called from device code
    ret = g_k_dev;
#else
    // Called from host code
    ret = g_k;
#endif
    assert(ret);
    return *ret;
}
}  // namespace

namespace coreneuron {
std::size_t nrnran123_instance_count() {
    return g_instance_count;
}

/* if one sets the global, one should reset all the stream sequences. */
uint32_t nrnran123_get_globalindex() {
    setup_global_state();
    return get_global_state().v[0];
}

CORENRN_HOST_DEVICE void nrnran123_getseq(nrnran123_State* s, uint32_t* seq, char* which) {
    *seq = s->c.v[0];
    *which = s->which_;
}

CORENRN_HOST_DEVICE void nrnran123_setseq(nrnran123_State* s, uint32_t seq, char which) {
    if (which > 3) {
        s->which_ = 0;
    } else {
        s->which_ = which;
    }
    s->c.v[0] = seq;
    s->r = philox4x32(s->c, get_global_state());
}

CORENRN_HOST_DEVICE void nrnran123_getids(nrnran123_State* s, uint32_t* id1, uint32_t* id2) {
    *id1 = s->c.v[2];
    *id2 = s->c.v[3];
}

CORENRN_HOST_DEVICE void nrnran123_getids3(nrnran123_State* s,
                                           uint32_t* id1,
                                           uint32_t* id2,
                                           uint32_t* id3) {
    *id3 = s->c.v[1];
    *id1 = s->c.v[2];
    *id2 = s->c.v[3];
}

CORENRN_HOST_DEVICE uint32_t nrnran123_ipick(nrnran123_State* s) {
    uint32_t rval;
    char which = s->which_;
    rval = s->r.v[which++];
    if (which > 3) {
        which = 0;
        s->c.v[0]++;
        s->r = philox4x32(s->c, get_global_state());
    }
    s->which_ = which;
    return rval;
}

CORENRN_HOST_DEVICE double nrnran123_dblpick(nrnran123_State* s) {
    return nrnran123_uint2dbl(nrnran123_ipick(s));
}

CORENRN_HOST_DEVICE double nrnran123_negexp(nrnran123_State* s) {
    /* min 2.3283064e-10 to max 22.18071 */
    return -std::log(nrnran123_dblpick(s));
}

/* at cost of a cached  value we could compute two at a time. */
CORENRN_HOST_DEVICE double nrnran123_normal(nrnran123_State* s) {
    double w, x, y;
    double u1, u2;

    do {
        u1 = nrnran123_dblpick(s);
        u2 = nrnran123_dblpick(s);
        u1 = 2. * u1 - 1.;
        u2 = 2. * u2 - 1.;
        w = (u1 * u1) + (u2 * u2);
    } while (w > 1);

    y = std::sqrt((-2. * log(w)) / w);
    x = u1 * y;
    return x;
}

CORENRN_HOST_DEVICE double nrnran123_uint2dbl(uint32_t u) {
    /* 0 to 2^32-1 transforms to double value in open (0,1) interval */
    /* min 2.3283064e-10 to max (1 - 2.3283064e-10) */
    return ((double) u + 1.0) * SHIFT32;
}

/* nrn123 streams are created from cpu launcher routine */
void nrnran123_set_globalindex(uint32_t gix) {
    setup_global_state();
    // If the global seed is changing then we shouldn't have any active streams.
    {
        std::lock_guard<OMP_Mutex> _{g_instance_count_mutex};
        if (g_instance_count != 0 && nrnmpi_myid == 0) {
            std::cout
                << "nrnran123_set_globalindex(" << gix
                << ") called when a non-zero number of Random123 streams (" << g_instance_count
                << ") were active. This is not safe, some streams will remember the old value ("
                << get_global_state().v[0] << ')' << std::endl;
        }
    }
    get_global_state().v[0] = gix;
}

/** @brief Allocate a new Random123 stream.
 *  @todo  It would be nicer if the API return type was
 *  std::unique_ptr<nrnran123_State, ...not specified...>, so we could use a
 *  custom allocator/deleter and avoid the (fragile) need for matching
 *  nrnran123_deletestream calls.
 */
nrnran123_State* nrnran123_newstream3(uint32_t id1,
                                      uint32_t id2,
                                      uint32_t id3,
                                      bool use_unified_memory) {
    // The `use_unified_memory` argument is an implementation detail to keep the
    // old behaviour that some Random123 streams that are known to only be used
    // from the CPU are allocated using new/delete instead of unified memory.
    // See OPENACC_EXCLUDED_FILES in coreneuron/CMakeLists.txt. If we dropped
    // this feature then we could always use coreneuron::unified_allocator.
#ifndef CORENEURON_ENABLE_GPU
    if (use_unified_memory) {
        throw std::runtime_error("Tried to use CUDA unified memory in a non-GPU build.");
    }
#endif
    nrnran123_State* s{nullptr};
    if (use_unified_memory) {
        s = coreneuron::allocate_unique<nrnran123_State>(
                coreneuron::unified_allocator<nrnran123_State>{})
                .release();
    } else {
        s = new nrnran123_State{};
    }
    s->c.v[0] = 0;
    s->c.v[1] = id3;
    s->c.v[2] = id1;
    s->c.v[3] = id2;
    nrnran123_setseq(s, 0, 0);
    {
        // TODO: can I assert something useful about the instance count going
        // back to zero anywhere? Or that it is zero when some operations happen?
        std::lock_guard<OMP_Mutex> _{g_instance_count_mutex};
        ++g_instance_count;
    }
    return s;
}

/* nrn123 streams are destroyed from cpu launcher routine */
void nrnran123_deletestream(nrnran123_State* s, bool use_unified_memory) {
#ifndef CORENEURON_ENABLE_GPU
    if (use_unified_memory) {
        throw std::runtime_error("Tried to use CUDA unified memory in a non-GPU build.");
    }
#endif
    {
        std::lock_guard<OMP_Mutex> _{g_instance_count_mutex};
        --g_instance_count;
    }
    if (use_unified_memory) {
        std::unique_ptr<nrnran123_State,
                        coreneuron::alloc_deleter<coreneuron::unified_allocator<nrnran123_State>>>
            _{s};
    } else {
        delete s;
    }
}
}  // namespace coreneuron
